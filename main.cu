#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <cstdio>
#include <iostream>
#include "src/main.hxx"

using namespace std;




#define REPEAT 5

template <class G, class H>
void runPagerank(const G& x, const H& xt, bool show) {
  vector<float> *init = nullptr;

  // Find pagerank using nvGraph.
  auto a1 = pagerankNvgraph(xt, init, {REPEAT});
  auto e1 = l1Norm(a1.ranks, a1.ranks);
  printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankNvgraph\n", a1.time, a1.iterations, e1);

  // Find pagerank without optimization.
  auto a2 = pagerankCuda(xt, init, {REPEAT});
  auto e2 = l1Norm(a2.ranks, a1.ranks);
  printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankCuda\n", a2.time, a2.iterations, e2);

  // Find pagerank skipping converged vertices with re-check.
  for (int skipCheck=2; skipCheck<16; skipCheck+=int(log2(skipCheck))) {
    auto a3 = pagerankCuda(xt, init, {REPEAT, skipCheck, 0});
    auto e3 = l1Norm(a3.ranks, a1.ranks);
    printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankCuda [skip-check=%d]\n", a3.time, a3.iterations, e3, skipCheck);
  }

  // Find pagerank skipping converged vertices after several turns.
  for (int skipAfter=2; skipAfter<64; skipAfter+=int(log2(skipAfter))) {
    auto a4 = pagerankCuda(xt, init, {REPEAT, 0, skipAfter});
    auto e4 = l1Norm(a4.ranks, a1.ranks);
    printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankCuda [skip-after=%d]\n", a4.time, a4.iterations, e4, skipAfter);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  bool  show = argc > 2;
  printf("Loading graph %s ...\n", file);
  auto x  = readMtx(file); println(x);
  auto xt = transposeWithDegree(x); print(xt); printf(" (transposeWithDegree)\n");
  runPagerank(x, xt, show);
  printf("\n");
  return 0;
}
