#include "hip/hip_runtime.h"
#include <vector>
#include <cstdio>
#include <iostream>
#include "src/main.hxx"

using namespace std;




#define REPEAT 5

template <class G, class H>
void runPagerank(const G& x, const H& xt, bool show) {
  vector<float> *init = nullptr;

  // Find pagerank using nvGraph.
  auto a1 = pagerankNvgraph(xt, init, {REPEAT});
  auto e1 = l1Norm(a1.ranks, a1.ranks);
  printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankNvgraph\n", a1.time, a1.iterations, e1);

  // Find CUDA based pagerank, adjust min-component-size from 1E+0 - 1E+8.
  for (int C=1, i=0; C<5*xt.order(); C*=i&1? 2:5, i++) {
    int minComponentSize = C;
    auto a2 = pagerankCuda(x, xt, init, {REPEAT, minComponentSize});
    auto e2 = l1Norm(a2.ranks, a1.ranks);
    printf("[%09.3f ms; %03d iters.] [%.4e err.] pagerankCuda [min-component-size=%d]\n", a2.time, a2.iterations, e2, minComponentSize);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  bool  show = argc > 2;
  printf("Loading graph %s ...\n", file);
  auto x  = readMtx(file); println(x);
  auto xt = transposeWithDegree(x); print(xt); printf(" (transposeWithDegree)\n");
  runPagerank(x, xt, show);
  printf("\n");
  return 0;
}
